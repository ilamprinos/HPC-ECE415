/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	1


#define CudaErrorCheck() \
	error = hipGetLastError(); \
	if (error != hipSuccess) { \
    		printf("Cuda error %s: %d: '%s'.\n", __FILE__, __LINE__, hipGetErrorString(error)); \
   		freeHostandDevice(h_OutputCPU, h_Buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input, d_Filter); \
   		hipDeviceReset(); \
   		exit(1); \
	}




void freeHostandDevice(float *h_OutputCPU, float *h_Buffer, float *h_Input, float *h_Filter, float *h_OutputGPU, float *d_OutputGPU, float *d_Buffer, float *d_Input, float *d_Filter) {

	free(h_OutputCPU); 
	free(h_Buffer); 
	free(h_Input); 
	free(h_Filter); 
	free(h_OutputGPU); 
	hipFree(d_OutputGPU); 
	hipFree(d_Buffer); 
	hipFree(d_Input); 
	hipFree(d_Filter);
}

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
  
  
                      
  for (y = filterR; y < (imageH - filterR); y++) {
    for (x = filterR; x < (imageW - filterR); x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
             
      }
      
      h_Dst[y * imageW + x] = sum;
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = filterR; y < (imageH - filterR); y++) {
    for (x = filterR; x < (imageW - filterR); x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
 
      }
      
      h_Dst[y * imageW + x] = sum;
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, 
                       int imageW, int imageH, int filterR) {

  int k, x, y;
                      
      float sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        sum += d_Src[y * imageW + d] * d_Filter[filterR - k];     
        
      }
      
      d_Dst[y * imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////
__global__  void convolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter,
    			   int imageW, int imageH, int filterR) {

  int k, x, y;
  
      float sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        sum += d_Src[d * imageW + x] * d_Filter[filterR - k];  
      }
      
      d_Dst[y * imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
    *h_OutputGPU,
    max_error;
    
    
    int imageW, padded_imageW;
    int imageH, padded_imageH;
    unsigned int i,j;
    hipError_t error;
    dim3 dimGrid, dimBlock;
    struct timespec  tv1, tv2;
    hipEvent_t start, stop;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;


    padded_imageW = 2*filter_radius + imageW;
    padded_imageH = 2*filter_radius + imageH;
    

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    
    
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(padded_imageW * padded_imageH * sizeof(float));
    h_Buffer    = (float *)malloc(padded_imageW * padded_imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(padded_imageW * padded_imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(padded_imageW * padded_imageH * sizeof(float));
    
    if( (h_Filter==NULL) || (h_Input==NULL) || (h_Buffer==NULL) || (h_OutputCPU==NULL) || (h_OutputGPU==NULL) ){
    	printf("Error while allocating host memory.\n");
    	exit(1);
    }
    

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.
    
    printf("Allocating and initializing device arrays...\n");
    hipMalloc( (void**)&d_Filter, FILTER_LENGTH * sizeof(float));
    hipMalloc( (void**)&d_Input, padded_imageW * padded_imageH * sizeof(float));
    hipMalloc( (void**)&d_Buffer, padded_imageW * padded_imageH * sizeof(float));
    hipMalloc( (void**)&d_OutputGPU,padded_imageW * padded_imageH * sizeof(float));    
  
    if( (d_Filter==NULL) || (d_Input==NULL) || (d_Buffer==NULL) || (d_OutputGPU==NULL) ){
    	printf("Error while allocating device memory.\n");
    	exit(1);
    }
    

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < padded_imageW; i++) {
    	for (j = 0; j < padded_imageH; j++){
            h_Input[i*padded_imageW+j] = 0;
        }
    }

    for (i = filter_radius; i < (padded_imageW - filter_radius); i++) {
    	for (j = filter_radius; j < (padded_imageH - filter_radius); j++){
            h_Input[i*padded_imageW+j] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
        }
    }
    
    
    if(imageW <= 32){
    	dimBlock.x = imageW;
   	dimBlock.y = imageH;
   	
   	dimGrid.x = 1;
   	dimGrid.y = 1;
    }
    
    else{
    
   	dimBlock.x = 32;
   	dimBlock.y = 32;
   	
   	dimGrid.x = imageW / 32;
   	dimGrid.y = imageH / 32;
    
    }
    
 
    hipEventCreate(&start);
    hipEventCreate(&stop);
     
    
    /**************************************   CPU computation   ******************************************/   

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, padded_imageW, padded_imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, padded_imageW, padded_imageH, filter_radius); // convolution kata sthles
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

    /**************************************   GPU computation   ******************************************/   
    
    
    printf("GPU computation...\n");
    
    
    hipEventRecord(start);
    
    hipMemset(d_Buffer, 0, padded_imageW * padded_imageW * sizeof(float));
    CudaErrorCheck();
    hipMemcpy( d_Filter, h_Filter, FILTER_LENGTH * sizeof(float) , hipMemcpyHostToDevice );
    CudaErrorCheck();
    hipMemcpy( d_Input, h_Input, padded_imageW * padded_imageH * sizeof(float) , hipMemcpyHostToDevice );
    CudaErrorCheck();
    
    convolutionRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, padded_imageW, padded_imageH, filter_radius);
    hipDeviceSynchronize();
    CudaErrorCheck();
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, padded_imageW, padded_imageH, filter_radius);
    hipDeviceSynchronize();
    CudaErrorCheck();
    
    hipMemcpy( h_OutputGPU, d_OutputGPU, padded_imageW * padded_imageH * sizeof(float) , hipMemcpyDeviceToHost );
    CudaErrorCheck(); 
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    
   
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    
    
    for (i = filter_radius; i < (padded_imageW - filter_radius); i++) {
    	for (j = filter_radius; j < (padded_imageH - filter_radius); j++){
            if( ABS(h_OutputGPU[i*padded_imageW + j] - h_OutputCPU[i*padded_imageW + j]) > max_error) {
		max_error = ABS(h_OutputGPU[i*padded_imageW + j] - h_OutputCPU[i*padded_imageW + j]);
    	}
        }
    }
    
    
    printf("Max absolute error: %f.\n", max_error);
    
    printf ("Total CPU time = %.9f seconds\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));

    printf("Total GPU time = %.9f seconds\n", elapsed / 1000);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // free all the allocated memory
    freeHostandDevice(h_OutputCPU, h_Buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input, d_Filter);
    
    
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
