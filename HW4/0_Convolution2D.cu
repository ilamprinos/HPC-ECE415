/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	1


#define CudaErrorCheck() \
	error = hipGetLastError(); \
	if (error != hipSuccess) { \
    		printf("Cuda error %s: %d: '%s'.\n", __FILE__, __LINE__, hipGetErrorString(error)); \
   		freeHostandDevice(h_OutputCPU, h_Buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input, d_Filter); \
   		hipDeviceReset(); \
   		exit(1); \
	}




void freeHostandDevice(double *h_OutputCPU, double *h_Buffer, double *h_Input, double *h_Filter, double *h_OutputGPU, double *d_OutputGPU, double *d_Buffer, double *d_Input, double *d_Filter) {

	free(h_OutputCPU); 
	free(h_Buffer); 
	free(h_Input); 
	free(h_Filter); 
	free(h_OutputGPU); 
	hipFree(d_OutputGPU); 
	hipFree(d_Buffer); 
	hipFree(d_Input); 
	hipFree(d_Filter);
}

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR) {

  int x, y, k;
  
  
                      
  for (y = filterR; y < (imageH - filterR); y++) {
    for (x = filterR; x < (imageW - filterR); x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {

        sum += h_Src[y * imageW + x + k] * h_Filter[filterR - k];
             
      }
      
      h_Dst[y * imageW + x] = sum;
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = filterR; y < (imageH - filterR); y++) {
    for (x = filterR; x < (imageW - filterR); x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {

        sum += h_Src[(y+k) * imageW + x] * h_Filter[filterR - k];
 
      }
      
      h_Dst[y * imageW + x] = sum;
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, double *d_Filter, int imageW, int imageH, int filterR) {

  int k, x, y;
                      
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;

      for (k = -filterR; k <= filterR; k++) {

        sum += d_Src[y * imageW +  x + k] * d_Filter[filterR - k];     
        
      }
      
      d_Dst[y * imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////
__global__  void convolutionColumnGPU(double *d_Dst, double *d_Src, double *d_Filter, int imageW, int imageH, int filterR) {

  int k, x, y;
  
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;

      for (k = -filterR; k <= filterR; k++) {

        sum += d_Src[(y + k) * imageW + x] * d_Filter[filterR - k];  
      }
      
      d_Dst[y * imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
    *h_OutputGPU,
    max_error;
    
    
    int imageW, padded_imageW;
    int imageH, padded_imageH;
    unsigned int i,j;
    hipError_t error;
    dim3 dimGrid, dimBlock;
    struct timespec  tv1, tv2;
    hipEvent_t start, stop;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;


    padded_imageW = 2*filter_radius + imageW;
    padded_imageH = 2*filter_radius + imageH;
    

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    
    
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    h_Buffer    = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    
    if( (h_Filter==NULL) || (h_Input==NULL) || (h_Buffer==NULL) || (h_OutputCPU==NULL) || (h_OutputGPU==NULL) ){
    	printf("Error while allocating host memory.\n");
    	exit(1);
    }
    

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.
    
    printf("Allocating and initializing device arrays...\n");
    hipMalloc( (void**)&d_Filter, FILTER_LENGTH * sizeof(double));
    hipMalloc( (void**)&d_Input, padded_imageW * padded_imageH * sizeof(double));
    hipMalloc( (void**)&d_Buffer, padded_imageW * padded_imageH * sizeof(double));
    hipMalloc( (void**)&d_OutputGPU,padded_imageW * padded_imageH * sizeof(double));    
  
    if( (d_Input==NULL) || (d_Buffer==NULL) || (d_OutputGPU==NULL) ){
    	printf("Error while allocating device memory.\n");
    	exit(1);
    }
    

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < padded_imageW; i++) {
    	for (j = 0; j < padded_imageH; j++){
            h_Input[i*padded_imageW+j] = 0;
        }
    }

    for (i = filter_radius; i < (padded_imageW - filter_radius); i++) {
    	for (j = filter_radius; j < (padded_imageH - filter_radius); j++){
            h_Input[i*padded_imageW+j] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
        }
    }
    
    
    if(imageW <= 32){
    	dimBlock.x = imageW;
   	dimBlock.y = imageH;
   	
   	dimGrid.x = 1;
   	dimGrid.y = 1;
    }
    
    else{
    
   	dimBlock.x = 32;
   	dimBlock.y = 32;
   	
   	dimGrid.x = imageW / 32;
   	dimGrid.y = imageH / 32;
    
    }
    
 
    hipEventCreate(&start);
    hipEventCreate(&stop);
     
    
    /**************************************   CPU computation   ******************************************/   

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, padded_imageW, padded_imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, padded_imageW, padded_imageH, filter_radius); // convolution kata sthles
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

    /**************************************   GPU computation   ******************************************/   
    
    
    printf("GPU computation...\n");
    
    
    hipEventRecord(start);
    
    hipMemset(d_Buffer, 0, padded_imageW * padded_imageW * sizeof(double));
    CudaErrorCheck();
    hipMemcpy( d_Filter, h_Filter, FILTER_LENGTH * sizeof(double) , hipMemcpyHostToDevice );
    CudaErrorCheck();
    hipMemcpy( d_Input, h_Input, padded_imageW * padded_imageH * sizeof(double) , hipMemcpyHostToDevice );
    CudaErrorCheck();
    
    convolutionRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, padded_imageW, padded_imageH, filter_radius);
    hipDeviceSynchronize();
    CudaErrorCheck();
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, padded_imageW, padded_imageH, filter_radius);
    hipDeviceSynchronize();
    CudaErrorCheck();
    
    hipMemcpy( h_OutputGPU, d_OutputGPU, padded_imageW * padded_imageH * sizeof(double) , hipMemcpyDeviceToHost );
    CudaErrorCheck(); 
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    
   
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    
    
    for (i = filter_radius; i < (padded_imageW - filter_radius); i++) {
    	for (j = filter_radius; j < (padded_imageH - filter_radius); j++){
            if( ABS(h_OutputGPU[i*padded_imageW + j] - h_OutputCPU[i*padded_imageW + j]) > max_error) {
		max_error = ABS(h_OutputGPU[i*padded_imageW + j] - h_OutputCPU[i*padded_imageW + j]);
    	}
        }
    }
    
    
    printf("Max absolute error: %f.\n", max_error);
    
    printf ("Total CPU time = %.9f seconds\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));

    printf("Total GPU time = %.9f seconds\n", elapsed / 1000);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // free all the allocated memory
    freeHostandDevice(h_OutputCPU, h_Buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input, d_Filter);
    
    
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
