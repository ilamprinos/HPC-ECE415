/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	1


#define CudaErrorCheck() \
	error = hipGetLastError(); \
	if (error != hipSuccess) { \
    		printf("Cuda error %s: %d: '%s'.\n", __FILE__, __LINE__, hipGetErrorString(error)); \
   		freeHostandDevice(h_OutputCPU, h_Buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input, d_Filter); \
   		hipDeviceReset(); \
   		exit(1); \
	}




void freeHostandDevice(double *h_OutputCPU, double *h_Buffer, double *h_Input, double *h_Filter, double *h_OutputGPU, double *d_OutputGPU, double *d_Buffer, double *d_Input, double *d_Filter) {

	free(h_OutputCPU); 
	free(h_Buffer); 
	free(h_Input); 
	free(h_Filter); 
	free(h_OutputGPU); 
	hipFree(d_OutputGPU); 
	hipFree(d_Buffer); 
	hipFree(d_Input); 
	hipFree(d_Filter);
}

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
  
  
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     
      }
      
      h_Dst[y * imageW + x] = sum;
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
      }
      
      h_Dst[y * imageW + x] = sum;
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, double *d_Filter, 
                       int imageW, int imageH, int filterR) {

  int k, x, y;
                      
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x;
      y = blockIdx.y*blockDim.y + threadIdx.y;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += d_Src[y * imageW + d] * d_Filter[filterR - k];
        }     
      }
      
      d_Dst[y * imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////
__global__  void convolutionColumnGPU(double *d_Dst, double *d_Src, double *d_Filter,
    			   int imageW, int imageH, int filterR) {

  int k, x, y;
  
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x;
      y = blockIdx.y*blockDim.y + threadIdx.y;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += d_Src[d * imageW + x] * d_Filter[filterR - k];
        }   
      }
      
      d_Dst[y * imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
    *h_OutputGPU,
    max_error=0;
    
    
    int imageW;
    int imageH;
    unsigned int i;
    hipError_t error;
    dim3 dimGrid, dimBlock;
    struct timespec  tv1, tv2;
    hipEvent_t start, stop;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    
    
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));
    
    if( (h_Filter==NULL) || (h_Input==NULL) || (h_Buffer==NULL) || (h_OutputCPU==NULL) || (h_OutputGPU==NULL) ){
    	printf("Error while allocating host memory.\n");
    	exit(1);
    }
    

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.
    
    printf("Allocating and initializing device arrays...\n");
    hipMalloc( (void**)&d_Filter, FILTER_LENGTH * sizeof(double));
    hipMalloc( (void**)&d_Input, imageW * imageH * sizeof(double));
    hipMalloc( (void**)&d_Buffer, imageW * imageH *sizeof(double));
    hipMalloc( (void**)&d_OutputGPU,imageW * imageH * sizeof(double));    
  
    if( (d_Filter==NULL) || (d_Input==NULL) || (d_Buffer==NULL) || (d_OutputGPU==NULL) ){
    	printf("Error while allocating device memory.\n");
    	exit(1);
    }
    

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }
    
    
    if(imageW <= 32){
    	dimBlock.x = imageW;
   	dimBlock.y = imageH;
   	
   	dimGrid.x = 1;
   	dimGrid.y = 1;
    }
    
    else{
    
   	dimBlock.x = 32;
   	dimBlock.y = 32;
   	
   	dimGrid.x = imageW / 32;
   	dimGrid.y = imageH / 32;
    
    }
    
 
    hipEventCreate(&start);
    hipEventCreate(&stop);
     
    
    /**************************************   CPU computation   ******************************************/   

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

    /**************************************   GPU computation   ******************************************/   
    
    
    printf("GPU computation...\n");
    
    
    hipEventRecord(start);
    
    hipMemcpy( d_Filter, h_Filter,FILTER_LENGTH * sizeof(double) , hipMemcpyHostToDevice );
    CudaErrorCheck();
    hipMemcpy( d_Input, h_Input,imageW * imageH * sizeof(double) , hipMemcpyHostToDevice );
    CudaErrorCheck();
    
    convolutionRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    CudaErrorCheck();
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    CudaErrorCheck();
    
    hipMemcpy( h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(double) , hipMemcpyDeviceToHost );
    CudaErrorCheck(); 
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    
   
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    
    for(i=0; i < imageW * imageH; i++){
    	if( ABS(h_OutputGPU[i] - h_OutputCPU[i]) > max_error) {
		max_error = ABS(h_OutputGPU[i] - h_OutputCPU[i]);
    	}
    }
    
    printf("Max absolute error: %f.\n", max_error);
    
    printf ("Total CPU time = %.9f seconds\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));

    printf("Total GPU time = %.9f seconds\n", elapsed / 1000);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // free all the allocated memory
    freeHostandDevice(h_OutputCPU, h_Buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input, d_Filter);
    
    
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
