/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define filterR 32
#define FILTER_LENGTH 	(2 * filterR + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.5

#define CudaErrorCheck() \
	error = hipGetLastError(); \
	if (error != hipSuccess) { \
    		printf("Cuda error %s: %d: '%s'.\n", __FILE__, __LINE__, hipGetErrorString(error)); \
   		freeHostandDevice(h_OutputCPU, h_Buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input); \
   		hipDeviceReset(); \
   		exit(1); \
	}

__constant__ double d_Filter[FILTER_LENGTH];


void freeHostandDevice(double *h_OutputCPU, double *h_Buffer, double *h_Input, double *h_Filter, double *h_OutputGPU, double *d_OutputGPU, double *d_Buffer, double *d_Input) {

	free(h_OutputCPU); 
	free(h_Buffer); 
	free(h_Input); 
	free(h_Filter); 
	free(h_OutputGPU); 
	hipFree(d_OutputGPU); 
	hipFree(d_Buffer); 
	hipFree(d_Input); 
}

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, int padded_imageH, int padded_imageW) {

  int x, y, k;
                      
  for (y = filterR; y < (padded_imageH - filterR); y++) {
    for (x = filterR; x < (padded_imageW - filterR); x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        sum += h_Src[y * padded_imageW + x + k] * h_Filter[filterR - k];       
      }
      
      h_Dst[y * padded_imageW + x] = sum;
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter, int padded_imageH, int padded_imageW) {

  int x, y, k;
  
  for (y = filterR; y < (padded_imageH - filterR); y++) {
    for (x = filterR; x < (padded_imageW - filterR); x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        sum += h_Src[(y + k) * padded_imageW + x] * h_Filter[filterR - k];
      }

      h_Dst[y * padded_imageW + x] = sum;
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////


__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, int imageW) {

  int k, x, y;
                      
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;

      for (k = -filterR; k <= filterR; k++) {
        sum += d_Src[y * imageW + x + k] * d_Filter[filterR - k];     
      }
      
      d_Dst[y * imageW + x] = sum;
}



__global__ void convolutionRowTiledGPU(double *d_Dst, double *d_Src, int padded_imageW, int shared_mem_size) {


      int k, x, y, tx, ty;
                      
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;
      tx = threadIdx.x;
      ty = threadIdx.y;
      extern __shared__ double s_Src[];
      
      
      for(int p=0; p < shared_mem_size; p+=blockDim.x){
      	s_Src[ty*shared_mem_size + tx + p] = d_Src[y*padded_imageW + p + x-filterR];
      }
      __syncthreads();

      for (k = -filterR; k <= filterR; k++) {
        sum += s_Src[ty*shared_mem_size + tx + k + filterR] * d_Filter[filterR - k];     
      }
      d_Dst[y * padded_imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////


__global__  void convolutionColumnGPU(double *d_Dst, double *d_Src, int imageW) {

  int k, x, y;
  
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;

      for (k = -filterR; k <= filterR; k++) {
        sum += d_Src[(y+k) * imageW + x] * d_Filter[filterR - k];  
      }
      
      d_Dst[y * imageW + x] = sum;
}

__global__  void convolutionColumnTiledGPU(double *d_Dst, double *d_Src, int padded_imageW, int shared_mem_size) {
      
      int k, x, y, tx, ty;
                      
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;
      tx = threadIdx.x;
      ty = threadIdx.y;
      extern __shared__ double s_Src[];
      
      
      for(int p=0; p < shared_mem_size; p+=blockDim.x){
      	s_Src[(ty + p)*blockDim.x + tx] = d_Src[(y-filterR + p)*padded_imageW + x];
      }
      __syncthreads();
      
      for (k = -filterR; k <= filterR; k++) {
        sum += s_Src[(ty + k + filterR)*blockDim.x + tx] * d_Filter[filterR - k];  
      }
      d_Dst[y * padded_imageW + x] = sum;
}
                      
  
     

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
    *h_OutputGPU,
    max_error;
    
    int imageW, padded_imageW, imageH, padded_imageH, shared_mem_size,tile_size;
    unsigned int i,j;
    hipError_t error;
    dim3 dimGrid, dimBlock;
    struct timespec  tv1, tv2;
    hipEvent_t start, stop;

	/*printf("Enter filter radius : ");
	scanf("%d", &filterR);*/

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;
    
    padded_imageH = 2*filterR + imageH;
    padded_imageW = 2*filterR + imageW;


    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    
    
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    h_Buffer    = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    
    if( (h_Filter==NULL) || (h_Input==NULL) || (h_Buffer==NULL) || (h_OutputCPU==NULL) || (h_OutputGPU==NULL) ){
    	printf("Error while allocating host memory.\n");
    	exit(1);
    }
    

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.
    
    printf("Allocating and initializing device arrays...\n");
    hipMalloc( (void**)&d_Input, padded_imageW * padded_imageH * sizeof(double));
    hipMalloc( (void**)&d_Buffer, padded_imageW * padded_imageH * sizeof(double));
    hipMalloc( (void**)&d_OutputGPU,padded_imageW * padded_imageH * sizeof(double));    
  
    if( (d_Input==NULL) || (d_Buffer==NULL) || (d_OutputGPU==NULL) ){
    	printf("Error while allocating device memory.\n");
    	exit(1);
    }
    

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < padded_imageW; i++) {
    	for (j = 0; j < padded_imageH; j++){
            h_Input[i*padded_imageW+j] = 0;
        }
    }

    for (i = filterR; i < (padded_imageW - filterR); i++) {
    	for (j = filterR; j < (padded_imageH - filterR); j++){
            h_Input[i*padded_imageW+j] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
        }
    }
    
    
    if(imageW <= 32){
    	dimBlock.x = imageW;
   	dimBlock.y = imageH;
   	
   	dimGrid.x = 1;
   	dimGrid.y = 1;
    }
    
    else{
    
   	dimBlock.x = 32;
   	dimBlock.y = 32;
   	
   	dimGrid.x = imageW / 32;
   	dimGrid.y = imageH / 32;
    
    }
    
 	shared_mem_size = 2*filterR + dimBlock.x;
 	tile_size = dimBlock.y*(dimBlock.x+2*filterR)*sizeof(double);

    hipEventCreate(&start);
    hipEventCreate(&stop);
     
    
    /**************************************   CPU computation   ******************************************/   

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter,padded_imageW, padded_imageH); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, padded_imageW, padded_imageH); // convolution kata sthles
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

    /**************************************   GPU computation   ******************************************/   
    
    
    printf("GPU computation...\n");
    
    
    hipEventRecord(start);
    
    hipMemset(d_Buffer, 0, padded_imageW * padded_imageW * sizeof(double));
    CudaErrorCheck();
    hipMemcpyToSymbol( HIP_SYMBOL(d_Filter), h_Filter, FILTER_LENGTH * sizeof(double) , 0, hipMemcpyHostToDevice );
    CudaErrorCheck();
    hipMemcpy( d_Input, h_Input, padded_imageW * padded_imageH * sizeof(double) , hipMemcpyHostToDevice );
    CudaErrorCheck();
    
    convolutionRowTiledGPU<<<dimGrid, dimBlock, tile_size>>>(d_Buffer, d_Input, padded_imageW, shared_mem_size);
    hipDeviceSynchronize();
    CudaErrorCheck();
    
    convolutionColumnTiledGPU<<<dimGrid, dimBlock, tile_size>>>(d_OutputGPU, d_Buffer, padded_imageW, shared_mem_size);
    hipDeviceSynchronize();
    CudaErrorCheck();
    
    hipMemcpy( h_OutputGPU, d_OutputGPU, padded_imageW * padded_imageH * sizeof(double) , hipMemcpyDeviceToHost );
    CudaErrorCheck(); 
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    
   
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    
    
    for (i = filterR; i < (padded_imageW - filterR); i++) {
    	for (j = filterR; j < (padded_imageH - filterR); j++){
            if( ABS(h_OutputGPU[i*padded_imageW + j] - h_OutputCPU[i*padded_imageW + j]) > max_error) {
		max_error = ABS(h_OutputGPU[i*padded_imageW + j] - h_OutputCPU[i*padded_imageW + j]);
    	}
        }
    }
    
    
    printf("Max absolute error: %f.\n", max_error);
    
    printf ("Total CPU time = %.9f seconds\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));

    printf("Total GPU time = %.9f seconds\n", elapsed / 1000);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // free all the allocated memory
    freeHostandDevice(h_OutputCPU, h_Buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input);
    
    
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
