/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//#define CPU_use
#define filterR 32
#define BLOCKING 512
#define padded_blocking (2 * filterR + BLOCKING) 
#define FILTER_LENGTH 	(2 * filterR + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.5

#define CudaErrorCheck() \
	error = hipGetLastError(); \
	if (error != hipSuccess) { \
    	printf("Cuda error %s: %d: '%s'.\n", __FILE__, __LINE__, hipGetErrorString(error)); \
   		freeHostandDevice(blocked_buffer, blocked_input, blocked_output, buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input); \
   		hipDeviceReset(); \
   		exit(1); \
	}

__constant__ double d_Filter[FILTER_LENGTH];


void freeifdef(double *h_OutputCPU, double *h_Buffer){

  free(h_OutputCPU); 
  free(h_Buffer); 
}

void freeHostandDevice(double *blocked_buffer, double *blocked_input, double *blocked_output, double *buffer, double *h_Input, double *h_Filter, double *h_OutputGPU, double *d_OutputGPU, double *d_Buffer, double *d_Input) {

	free(blocked_buffer); 
	free(blocked_input); 
  free(blocked_output); 
  free(buffer); 
	free(h_Input); 
	free(h_Filter); 
	free(h_OutputGPU); 
	hipFree(d_OutputGPU); 
	hipFree(d_Buffer); 
	hipFree(d_Input); 
}

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, int padded_imageH, int padded_imageW) {

  int x, y, k;
                      
  for (y = filterR; y < (padded_imageH - filterR); y++) {
    for (x = filterR; x < (padded_imageW - filterR); x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        sum += h_Src[y * padded_imageW + x + k] * h_Filter[filterR - k];       
      }
      
      h_Dst[y * padded_imageW + x] = sum;
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (CPU)
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter, int padded_imageH, int padded_imageW) {

  int x, y, k;
  
  for (y = filterR; y < (padded_imageH - filterR); y++) {
    for (x = filterR; x < (padded_imageW - filterR); x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        sum += h_Src[(y + k) * padded_imageW + x] * h_Filter[filterR - k];
      }

      h_Dst[y * padded_imageW + x] = sum;
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////


__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, int imageW) {

  int k, x, y;
                      
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;

      for (k = -filterR; k <= filterR; k++) {
        sum += d_Src[y * imageW + x + k] * d_Filter[filterR - k];     
      }
      
      d_Dst[y * imageW + x] = sum;
}



__global__ void convolutionRowTiledGPU(double *d_Dst, double *d_Src, int padded_imageW, int shared_mem_size) {


      int k, x, y, tx, ty;
                      
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;
      tx = threadIdx.x;
      ty = threadIdx.y;
      extern __shared__ double s_Src[];
      
      //int shared_mem_size = blockDim.x + 2*filterR; 
      
      for(int p=0; p < shared_mem_size; p+=blockDim.x){
      	s_Src[ty*shared_mem_size + tx + p] = d_Src[y*padded_imageW + p + x-filterR];
      }
      __syncthreads();

      for (k = -filterR; k <= filterR; k++) {
        sum += s_Src[ty*shared_mem_size + tx + k + filterR] * d_Filter[filterR - k];     
      }
      d_Dst[y * padded_imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter (GPU)
////////////////////////////////////////////////////////////////////////////////


__global__  void convolutionColumnGPU(double *d_Dst, double *d_Src, int imageW) {

  int k, x, y;
  
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;

      for (k = -filterR; k <= filterR; k++) {
        sum += d_Src[(y+k) * imageW + x] * d_Filter[filterR - k];  
      }
      
      d_Dst[y * imageW + x] = sum;
}

__global__  void convolutionColumnTiledGPU(double *d_Dst, double *d_Src, int padded_imageW, int shared_mem_size) {
      
      int k, x, y, tx, ty;
                      
      double sum = 0;
      x = blockIdx.x*blockDim.x + threadIdx.x + filterR;
      y = blockIdx.y*blockDim.y + threadIdx.y + filterR;
      tx = threadIdx.x;
      ty = threadIdx.y;
      extern __shared__ double s_Src[];
      
     // int shared_mem_size = blockDim.x + 2*filterR; 
      
      for(int p=0; p < shared_mem_size; p+=blockDim.x){
      	s_Src[(ty + p)*blockDim.x + tx] = d_Src[(y-filterR + p)*padded_imageW + x];
      }
      __syncthreads();
      
      for (k = -filterR; k <= filterR; k++) {
        sum += s_Src[(ty + k + filterR)*blockDim.x + tx] * d_Filter[filterR - k];  
      }
      d_Dst[y * padded_imageW + x] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
    *h_OutputGPU,
    *blocked_buffer,
    *blocked_input,
    *blocked_output,
    *buffer;

#ifdef CPU_use
    double *h_Buffer, *h_OutputCPU, max_error=0;
    struct timespec  tv1, tv2;
#endif

    long int imageW, padded_imageW, imageH, padded_imageH, shared_mem_size;
    unsigned int i,j;
    hipError_t error;
    dim3 dimGrid, dimBlock;
    hipEvent_t start, stop;
	/*printf("Enter filter radius : ");
	scanf("%d", &filterR);*/

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%ld", &imageW);
    imageH = imageW;
    
    padded_imageH = 2*filterR + imageH;
    padded_imageW = 2*filterR + imageW;


    printf("Image Width x Height = %li x %li\n\n", imageW, imageH);
    
    
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
#ifdef CPU_use
    h_Buffer    = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
#endif
    h_OutputGPU = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));
    blocked_buffer = (double *)malloc(padded_blocking * padded_blocking * sizeof(double));
    blocked_input = (double *)malloc(padded_blocking * padded_blocking * sizeof(double));
    blocked_output = (double *)malloc(padded_blocking * padded_blocking * sizeof(double));
    buffer = (double *)malloc(padded_imageW * padded_imageH * sizeof(double));

    if((h_Filter==NULL) || (h_Input==NULL) || (buffer==NULL) || (blocked_output==NULL) || (h_OutputGPU==NULL) || (blocked_input==NULL) || (blocked_buffer==NULL)){
    	printf("Error while allocating host memory.\n");
    	exit(1);
    }

#ifdef CPU_use  
    if((h_Buffer==NULL) || (h_OutputCPU==NULL)){
      printf("Error while allocating host memory.\n");
      exit(1);
    }
#endif
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.
    
    printf("Allocating and initializing device arrays...\n");
    hipMalloc( (void**)&d_Input, padded_blocking * padded_blocking * sizeof(double));
    hipMalloc( (void**)&d_Buffer, padded_blocking * padded_blocking * sizeof(double));
    hipMalloc( (void**)&d_OutputGPU,padded_blocking * padded_blocking * sizeof(double));    
  
    if( (d_Input==NULL) || (d_Buffer==NULL) || (d_OutputGPU==NULL) ){
    	printf("Error while allocating device memory.\n");
    	exit(1);
    }
    

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < padded_imageW; i++) {
    	for (j = 0; j < padded_imageH; j++){
            h_Input[i*padded_imageW+j] = 0;
        }
    }

    for (i = filterR; i < (padded_imageW - filterR); i++) {
    	for (j = filterR; j < (padded_imageH - filterR); j++){
            h_Input[i*padded_imageW+j] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
        }
    }
    
    
    if(imageW <= 32){
      dimBlock.x = imageW;
    	dimBlock.y = imageH;
   	
   	  dimGrid.x = 1;
   	  dimGrid.y = 1;
    }
    
    else{
    
   	  dimBlock.x = 32;
   	  dimBlock.y = 32;
   	
     	dimGrid.x = BLOCKING / 32;
   	  dimGrid.y = BLOCKING / 32;
    
    }

    shared_mem_size = 2*filterR + dimBlock.x;
    
 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    /**************************************   CPU computation   ******************************************/   
#ifdef CPU_use
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter,padded_imageW, padded_imageH); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, padded_imageW, padded_imageH); // convolution kata sthles
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
#endif
    /**************************************   GPU computation   ******************************************/   
    
    hipMemset(d_Buffer, 0, padded_blocking * padded_blocking * sizeof(double));
    CudaErrorCheck();
    hipMemset(d_OutputGPU, 0, padded_blocking * padded_blocking * sizeof(double));
    CudaErrorCheck();

    printf("GPU computation...\n");

    hipEventRecord(start);
    
    hipMemcpyToSymbol( HIP_SYMBOL(d_Filter), h_Filter, FILTER_LENGTH * sizeof(double) , 0, hipMemcpyHostToDevice );
    CudaErrorCheck();
    //cudaMemcpy( d_Input, h_Input, padded_imageW * padded_imageH * sizeof(double) , cudaMemcpyHostToDevice );
    //CudaErrorCheck();
    

    for(int y=0; y < imageW/BLOCKING; y++){
      for(int x=0; x < imageW/BLOCKING; x++){

        for(i=0; i < padded_blocking; i++){
          for(j=0; j < padded_blocking; j++){
            blocked_input[i*padded_blocking + j] = h_Input[(i + y*BLOCKING)*padded_imageW + BLOCKING*x + j];
          }
        }

        hipMemcpy(d_Input, blocked_input, padded_blocking*padded_blocking*sizeof(double), hipMemcpyHostToDevice);
        CudaErrorCheck();
        convolutionRowTiledGPU<<<dimGrid, dimBlock, dimBlock.y*(dimBlock.x+2*filterR)*sizeof(double)>>>(d_Buffer, d_Input, padded_blocking, shared_mem_size);
        hipDeviceSynchronize();
        CudaErrorCheck();
        hipMemcpy(blocked_buffer, d_Buffer, padded_blocking*padded_blocking*sizeof(double), hipMemcpyDeviceToHost);
        CudaErrorCheck();

        for(i=0; i < BLOCKING; i++){
          for(j=0; j < BLOCKING; j++){
            buffer[(i + filterR + y*BLOCKING)*padded_imageW + j + filterR + x*BLOCKING ] = blocked_buffer[(i+filterR)*padded_blocking + j +filterR];
          }
        }
      }
    }

    for(int y=0; y < imageW/BLOCKING; y++){
      for(int x=0; x < imageW/BLOCKING; x++){

        for(i=0; i < padded_blocking; i++){
          for(j=0; j < padded_blocking; j++){
            blocked_buffer[i*padded_blocking + j] =  buffer[(y*BLOCKING + i)*padded_imageW + BLOCKING*x + j];
          }
        }
        
        hipMemcpy(d_Buffer, blocked_buffer, padded_blocking*padded_blocking*sizeof(double), hipMemcpyHostToDevice);
        CudaErrorCheck();
        convolutionColumnTiledGPU<<<dimGrid, dimBlock, dimBlock.y*(dimBlock.x+2*filterR)*sizeof(double)>>>(d_OutputGPU, d_Buffer, padded_blocking, shared_mem_size);
        hipDeviceSynchronize();
        CudaErrorCheck();
        hipMemcpy(blocked_output, d_OutputGPU, padded_blocking*padded_blocking*sizeof(double), hipMemcpyDeviceToHost);
        CudaErrorCheck();

        for(i=0; i <  BLOCKING; i++){
          for(j=0; j < BLOCKING; j++){
            h_OutputGPU[(y*BLOCKING + i + filterR)*padded_imageW + j + filterR + x*BLOCKING] = blocked_output[(i+filterR)*padded_blocking + j +filterR];
          }
        }
      }
    } 
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    
   
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    
#ifdef CPU_use
    for (i = filterR; i < (imageW + filterR); i++) {
    	for (j = filterR; j < (imageH + filterR); j++){
        if( ABS(h_OutputGPU[i*padded_imageW + j] - h_OutputCPU[i*padded_imageW + j]) > max_error) {
		      max_error = ABS(h_OutputGPU[i*padded_imageW + j] - h_OutputCPU[i*padded_imageW + j]);
    	  }
      }
    }
    
    
    printf("Max absolute error: %f.\n", max_error);
    
    printf ("Total CPU time = %.9f seconds\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));
#endif
    printf("Total GPU time = %.9f seconds\n", elapsed / 1000);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // free all the allocated memory
    freeHostandDevice(blocked_buffer, blocked_input, blocked_output, buffer, h_Input, h_Filter, h_OutputGPU, d_OutputGPU, d_Buffer, d_Input); 

#ifdef CPU_use
    freeifdef(h_OutputCPU, h_Buffer); 
#endif 
    
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
